#ifndef _VECTOR_DOT_PRODUCT_KERNEL_H_
#define _VECTOR_DOT_PRODUCT_KERNEL_H_


#define BLOCK_SIZE 256
#define GRID_SIZE 240

/* prototypes */
__device__ void lock(int *mutex);
__device__ void unlock(int *mutex);

__global__ void vector_dot_product_kernel( float *A, float *B, float *C, unsigned int numElements, int *mutex) {

	__shared__ float thread_sums[ BLOCK_SIZE ];

	/* thread ID and stride lengths (for coalescing memory) */
	unsigned int tID = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int stride_length = blockDim.x * gridDim.x;

	/* initialize local thread sum and starting location for thread*/
	float local_thread_sum = 0.0f;
	unsigned int i = tID;

	/* perform multiplication and add stride_length continuously until max number of elements reached -->*/
	while( i < numElements ) {

		/* multiply, increment by stride */
		local_thread_sum += A[i] * B[i];
		i += stride_length;
	}

	/* Put thread sum in shared mem accessible to all thread blocks */
	thread_sums[threadIdx.x] = local_thread_sum;
	__syncthreads();


	/* REDUCTION -- Reduce thread sums on a per-block basis (so result in one sum per block) */
	i = BLOCK_SIZE / 2; 	
	while ( i != 0 ) {

		/* threads where i < 0 are threads on the second "half" which don't need to execute */
		if ( threadIdx.x < i ) {

			/* sum the calculating threads partial value with its second "half" counterpart */
			thread_sums[threadIdx.x] += thread_sums[ threadIdx.x + i ];
		}
		__syncthreads();

		/* reduces the threads by 2 each iteration */
		i = i / 2;
	}

	/* first thread in each block adds block-wide value to global mem location*/
	if (threadIdx.x == 0) {
		// define a lock
		lock(mutex);
		// add sums to global memory which is my critical section
		C[0] += thread_sums[0] ;
		// unlock the mutex/critical section
		unlock(mutex);
	}
}

__device__ void lock(int *mutex){
       	// if my mutex is 0 swap and set it to 1 indication locking of critical section
	while(atomicCAS(mutex, 0, 1) != 0);
}

/* Using exchange to release mutex. */
__device__ void unlock(int *mutex)
{      // perform an atomic exchange in which the pointer of the mutex now becomes 0	
       atomicExch(mutex, 0);
}



#endif // #ifndef _VECTOR_DOT_PRODUCT_KERNEL_H
